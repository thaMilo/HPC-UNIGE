#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>
#include <complex>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <sys/time.h>

#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)

#define STEP ((double)RATIO_X / WIDTH)

#define ITERATIONS 1000

using namespace std;

float time_diff(struct timeval *start, struct timeval *end) {
  	return (end->tv_sec - start->tv_sec) + 1e-6 * (end->tv_usec - start->tv_usec);
}

__device__ int mandelbrot(int col, int row, double step, int minX, int minY, int iterations)
{
	hipDoubleComplex c = make_hipDoubleComplex(col * step + minX, row * step + minY);
    hipDoubleComplex z = make_hipDoubleComplex(0, 0);

    int count = 0;
    while (hipCabs(z) < 2 && count < iterations)
    {
        z = hipCadd(hipCmul(z, z), c);
        count++;
    }

   	return (count < iterations) ? count : 0;
}

__global__ void generateMandelbrotSet(int *image, double step, int minX, int minY, int iterations)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

	int index = row * WIDTH + col;

	image[index] = mandelbrot(col, row, step, minX, minY, iterations);
}

int main(int argc, char *argv[])
{
	int threads = 1;
	if (argc < 2)
	{
		cout << "Please specify the output file as a parameter." << endl;
		return -1;
	}

	if (argc > 2)
	{
		threads = atoi(argv[2]);
	}

	cout << "Threads: " << threads << endl;
	cout << "Resolution: " << RESOLUTION << endl;

	int *image;
	int *d_image;
	size_t img_size = WIDTH * HEIGHT * sizeof(int);
	
	struct timeval start, stop, op_start, op_stop;
	hipEvent_t startEvent, stopEvent, startOpEvent, stopOpEvent;
	
	hipEventCreate(&startEvent); hipEventCreate(&stopEvent);
	hipEventCreate(&startOpEvent); hipEventCreate(&stopOpEvent);

	gettimeofday(&start, NULL);
	hipEventRecord(startEvent);

	hipMalloc(&d_image, img_size);

	// Allocate memory for the image on the host
	image = new int[HEIGHT * WIDTH];

	// Initialize all pixels to -1
	fill_n(image, WIDTH * HEIGHT, -1);

	// Copy the image to the device
	hipMemcpy(d_image, image, img_size, hipMemcpyHostToDevice);

	dim3 blockSize(threads, threads);
	dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

	//auto start = chrono::steady_clock::now();

	gettimeofday(&op_start, NULL);
	hipEventRecord(startOpEvent);
	generateMandelbrotSet<<<gridSize, blockSize>>>(d_image, STEP, MIN_X, MIN_Y, ITERATIONS);
	gettimeofday(&op_stop, NULL);
	hipEventRecord(stopOpEvent);
	hipEventSynchronize(stopOpEvent);

	// Copy the image back to the host
	hipMemcpy(image, d_image, img_size, hipMemcpyDeviceToHost);

	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent);
	gettimeofday(&stop, NULL);

	// Check if all pixels were computed
	/*
	bool allPixelsComputed = true;
	for (int i = 0; i < HEIGHT * WIDTH; i++)
	{
		if (image[i] == -1)
		{ // replace -1 with the value you use for uninitialized pixels
			allPixelsComputed = false;
			break;
		}
	}

	if (allPixelsComputed)
	{
		cout << "All pixels were computed." << endl;
	}
	else
	{
		cout << "Not all pixels were computed." << endl;
	}
	*/

	printf("Time spent: %0.8f ms\n", time_diff(&start, &stop)*1000);
  	printf("Op_Time spent: %0.8f ms\n", time_diff(&op_start, &op_stop)*1000);

	// Print time
	cout << "Time of operations: ";
	float time, timeOp;
	hipEventElapsedTime(&timeOp, startOpEvent, stopOpEvent);
	cout << fixed << setprecision(2) << timeOp << " ms" << endl;

	cout << "Time total: ";
	hipEventElapsedTime(&time, startEvent, stopEvent);
	cout << fixed << setprecision(2) << time << " ms" << endl;

	/*
	auto end = chrono::steady_clock::now();
	cout << "Time elapsed: "
		 << fixed << setprecision(2)
		 << chrono::duration<double>(end - start).count()
		 << " seconds." << endl;
	*/

	ofstream matrix_out(argv[1], ios::trunc);
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -2;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << image[row * WIDTH + col];
			if (col < WIDTH - 1)
				matrix_out << ',';
		}
		if (row < HEIGHT - 1)
			matrix_out << '\n';
	}
	matrix_out.close();

	delete[] image;
	hipFree(d_image);

	return 0;
}