#include <iostream>
#include <fstream>
#include <chrono>
#include <complex>
#include <iomanip>
#include <hip/hip_runtime.h>

#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

#define ITERATIONS 1000

using namespace std;

__device__ int mandelbrot(int col, int row, double step, int minX, int minY, int iterations)
{
	const complex<double> c(minX + col * step, minY + row * step);
	complex<double> z(0, 0);

	int count = 0;
	while (abs(z) < 2 && count <= iterations)
	{
		z = pow(z, 2) + c;
		count++;
	}

	if (count < iterations)
	{
		return count;
	}
	else
	{
		return 0;
	}
}

__global__ void generateMandelbrotSet(int *image, double step, int minX, int minY, int iterations, int WIDTH, int HEIGHT)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col >= WIDTH || row >= HEIGHT)
		return;

	int index = row * WIDTH + col;

	image[index] = mandelbrot(col, row, step, minX, minY, iterations);
}

int main(int argc, char **argv)
{
	int RESOLUTION = 1000;
	int threads = 2;

	if (argc < 2)
	{
		cout << "Please specify the output file as a parameter." << endl;
		return -1;
	} else {
		if (argc == 3)
		{
			RESOLUTION = atoi(argv[2]);
			threads = atoi(argv[3]);
		}
	}

	cout << "Resolution: " << RESOLUTION << endl;

	int WIDTH = (RATIO_X * RESOLUTION);
	int HEIGHT = (RATIO_Y * RESOLUTION);

	int STEP = ((double)RATIO_X / WIDTH);

	int *image;
	int *d_image;
	size_t img_size = WIDTH * HEIGHT * sizeof(int);

	hipMalloc(&d_image, img_size);

	// Allocate memory for the image on the host
	image = new int[HEIGHT * WIDTH];

	// Initialize all pixels to -1
	fill_n(image, WIDTH * HEIGHT, -1);

	// Copy the image to the device
	hipMemcpy(d_image, image, img_size, hipMemcpyHostToDevice);

	dim3 blockSize(threads, threads);
	dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);

	auto start = chrono::steady_clock::now();

	generateMandelbrotSet<<<gridSize, blockSize>>>(d_image, STEP, MIN_X, MIN_Y, ITERATIONS, WIDTH, HEIGHT);

	// Copy the image back to the host
	hipMemcpy(image, d_image, img_size, hipMemcpyDeviceToHost);

	// Check if all pixels were computed
	bool allPixelsComputed = true;
	for (int i = 0; i < HEIGHT * WIDTH; i++)
	{
		if (image[i] == -1)
		{ // replace -1 with the value you use for uninitialized pixels
			allPixelsComputed = false;
			break;
		}
	}

	if (allPixelsComputed)
	{
		cout << "All pixels were computed." << endl;
	}
	else
	{
		cout << "Not all pixels were computed." << endl;
	}

	auto end = chrono::steady_clock::now();
	cout << "Time elapsed: "
		 << fixed << setprecision(2)
		 << chrono::duration<double>(end - start).count()
		 << " seconds." << endl;

	ofstream matrix_out(argv[1], ios::trunc);
	if (!matrix_out.is_open())
	{
		cout << "Unable to open file." << endl;
		return -2;
	}

	for (int row = 0; row < HEIGHT; row++)
	{
		for (int col = 0; col < WIDTH; col++)
		{
			matrix_out << image[row * WIDTH + col];
			if (col < WIDTH - 1)
				matrix_out << ',';
		}
		if (row < HEIGHT - 1)
			matrix_out << '\n';
	}
	matrix_out.close();

	delete[] image;
	hipFree(d_image);

	return 0;
}